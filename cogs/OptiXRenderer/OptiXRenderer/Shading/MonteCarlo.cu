#include "hip/hip_runtime.h"
// OptiX path tracing ray generation and miss program.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Shading/ShadingModels/DefaultShading.h>
#include <OptiXRenderer/Shading/ShadingModels/LambertShading.h>
#include <OptiXRenderer/Shading/LightSources/LightImpl.h>
#include <OptiXRenderer/TBN.h>
#include <OptiXRenderer/Types.h>

#include <optix.h>

using namespace optix;
using namespace OptiXRenderer;
using namespace OptiXRenderer::Shading::ShadingModels;

// Ray parameters.
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(MonteCarloPRD, monte_carlo_PRD, rtPayload, );

// Scene parameters.
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );
rtBuffer<Light, 1> g_lights;
rtDeclareVariable(int, g_light_count, , );
rtDeclareVariable(int, g_max_bounce_count, , );
rtDeclareVariable(int, g_accumulations, , );

// Material parameters.
rtBuffer<Material, 1> g_materials;
rtDeclareVariable(int, material_index, , );

//----------------------------------------------------------------------------
// Closest hit program for monte carlo sampling rays.
//----------------------------------------------------------------------------

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Variables used for split screen debugging.
rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>  g_accumulation_buffer;

//-----------------------------------------------------------------------------
// Light sampling.
//-----------------------------------------------------------------------------

// Sample a single light source, evaluates the material's response to the light and 
// stores the combined response in the light source's radiance member.
__inline_dev__ LightSample sample_single_light(const DefaultShading& material, const TBN& world_shading_tbn) {
    int light_index = min(g_light_count - 1, int(monte_carlo_PRD.rng.sample1f() * g_light_count));
    const Light& light = g_lights[light_index];
    LightSample light_sample = LightSources::sample_radiance(light, monte_carlo_PRD.position, monte_carlo_PRD.rng.sample2f());
    light_sample.radiance *= g_light_count; // Scale up radiance to account for only sampling one light.

    // Inline the material response into the light sample's radiance.
    const float3 shading_light_direction = world_shading_tbn * light_sample.direction_to_light;
    const float3 bsdf_response = material.evaluate(monte_carlo_PRD.direction, shading_light_direction);// TODO Extend material and BRDFs with methods for evaluating contribution and PDF at the same time.
    light_sample.radiance *= bsdf_response;

    float N_dot_L = dot(world_shading_tbn.get_normal(), light_sample.direction_to_light);
    light_sample.radiance *= abs(N_dot_L) / light_sample.PDF;

    // Apply MIS weights if the light isn't a delta function and if a new material ray will be spawned, i.e. it isn't the final bounce.
    bool delta_light = LightSources::is_delta_light(light, monte_carlo_PRD.position);
    bool apply_MIS = !delta_light && monte_carlo_PRD.bounces < g_max_bounce_count;
    if (apply_MIS) {
        float bsdf_PDF = material.PDF(monte_carlo_PRD.direction, shading_light_direction);
        float mis_weight = RNG::power_heuristic(light_sample.PDF, bsdf_PDF); // TODO Check if the BSDF material PDF is valid. If it isn't we then disable MIS intirely? Or set contribution to black?

        light_sample.radiance *= mis_weight;
    }

    return light_sample;
}

// Take multiple light samples and from that set pick one based on the contribution of the light scaled by the material.
// Basic Resampled importance sampling: http://scholarsarchive.byu.edu/cgi/viewcontent.cgi?article=1662&context=etd.
__inline_dev__ LightSample reestimated_light_samples(const DefaultShading& material, const TBN& world_shading_tbn, int samples) {
    LightSample light_sample = sample_single_light(material, world_shading_tbn);
    for (int s = 1; s < samples; ++s) {
        LightSample new_light_sample = sample_single_light(material, world_shading_tbn);
        float light_weight = average(light_sample.radiance);
        float new_light_weight = average(new_light_sample.radiance);
        float new_light_probability = new_light_weight / (light_weight + new_light_weight);
        if (monte_carlo_PRD.rng.sample1f() < new_light_probability) {
            light_sample = new_light_sample;
            light_sample.radiance /= new_light_probability;
        } else
            light_sample.radiance /= 1.0f - new_light_probability;
    }
    light_sample.radiance /= samples;

    // NOTE If we want to use the accumlated path PDF later for filtering or firefly removal, then it's possible that we'd get better results by adjusting the PDF instead of just the total contribution.

    return light_sample;
}

//-----------------------------------------------------------------------------
// Closest hit integrators.
//-----------------------------------------------------------------------------

__inline_dev__ void closest_hit_not_MIS() {
    // const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 forward_shading_normal = -dot(world_shading_normal, ray.direction) >= 0.0f ? world_shading_normal : -world_shading_normal;

    const TBN world_shading_tbn = TBN(forward_shading_normal);

    // Store intersection point and wo in PRD.
    monte_carlo_PRD.position = ray.direction * t_hit + ray.origin;
    monte_carlo_PRD.direction = world_shading_tbn * -ray.direction;

    const Material& material_parameter = g_materials[material_index];
    const DefaultShading material = DefaultShading(material_parameter, texcoord);

    // Sample light sources.
    // TODO Use RIS light sampling here as well. But wait until I have a scene with multiple area light sources.
    for (int i = 0; i < g_light_count; ++i) {
        const Light& light = g_lights[i];
        LightSample light_sample = LightSources::sample_radiance(light, monte_carlo_PRD.position, monte_carlo_PRD.rng.sample2f());
        float N_dot_L = dot(world_shading_tbn.get_normal(), light_sample.direction_to_light);
        light_sample.radiance *= abs(N_dot_L) / light_sample.PDF;

        // Inline the material response into the light sample's contribution.
        const float3 shading_light_direction = world_shading_tbn * light_sample.direction_to_light;
        const float3 bsdf_response = material.evaluate(monte_carlo_PRD.direction, shading_light_direction);// TODO Extend material and BRDFs with methods for evaluating contribution and PDF at the same time.
        light_sample.radiance *= bsdf_response;

        if (light_sample.radiance.x > 0.0f || light_sample.radiance.y > 0.0f || light_sample.radiance.z > 0.0f) {
            ShadowPRD shadow_PRD = { light_sample.radiance };
            Ray shadow_ray(monte_carlo_PRD.position, light_sample.direction_to_light, unsigned int(RayTypes::Shadow), g_scene_epsilon, light_sample.distance - g_scene_epsilon);
            rtTrace(g_scene_root, shadow_ray, shadow_PRD);

            float3 radiance = monte_carlo_PRD.throughput * shadow_PRD.attenuation;
            monte_carlo_PRD.radiance += clamp_light_contribution_by_pdf(radiance, monte_carlo_PRD.clamped_path_PDF, g_accumulations);
        }
    }

    // Sample BSDF.
    BSDFSample bsdf_sample = material.sample_all(monte_carlo_PRD.direction, monte_carlo_PRD.rng.sample3f());
    monte_carlo_PRD.direction = bsdf_sample.direction * world_shading_tbn;
    monte_carlo_PRD.bsdf_MIS_PDF = 0.0f; // bsdf_sample.PDF;
    monte_carlo_PRD.path_PDF *= bsdf_sample.PDF;
    monte_carlo_PRD.clamped_path_PDF *= fminf(bsdf_sample.PDF, 1.0f);
    if (!is_PDF_valid(bsdf_sample.PDF))
        monte_carlo_PRD.throughput = make_float3(0.0f);
    else
        monte_carlo_PRD.throughput *= bsdf_sample.weight * (abs(bsdf_sample.direction.z) / bsdf_sample.PDF); // f * ||cos(theta)|| / pdf
    monte_carlo_PRD.bounces += 1u;
}

__inline_dev__ void closest_hit_MIS() {
    // const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 forward_shading_normal = -dot(world_shading_normal, ray.direction) >= 0.0f ? world_shading_normal : -world_shading_normal;

    const TBN world_shading_tbn = TBN(forward_shading_normal);

    // Store intersection point and wo in PRD.
    monte_carlo_PRD.position = ray.direction * t_hit + ray.origin;
    monte_carlo_PRD.direction = world_shading_tbn * -ray.direction;

    const Material& material_parameter = g_materials[material_index];
    const DefaultShading material = DefaultShading(material_parameter, texcoord);

    // Sample a light source.
    if (g_light_count != 0) {
        const LightSample light_sample = reestimated_light_samples(material, world_shading_tbn, 4);

        if (light_sample.radiance.x > 0.0f || light_sample.radiance.y > 0.0f || light_sample.radiance.z > 0.0f) {
            ShadowPRD shadow_PRD = { light_sample.radiance };
            Ray shadow_ray(monte_carlo_PRD.position, light_sample.direction_to_light, unsigned int(RayTypes::Shadow), g_scene_epsilon, light_sample.distance - g_scene_epsilon);
            rtTrace(g_scene_root, shadow_ray, shadow_PRD);

            float3 radiance = monte_carlo_PRD.throughput * shadow_PRD.attenuation;
            monte_carlo_PRD.radiance += clamp_light_contribution_by_pdf(radiance, monte_carlo_PRD.clamped_path_PDF, g_accumulations);
        }
    }

    // Sample BSDF.
    BSDFSample bsdf_sample = material.sample_all(monte_carlo_PRD.direction, monte_carlo_PRD.rng.sample3f());
    monte_carlo_PRD.direction = bsdf_sample.direction * world_shading_tbn;
    monte_carlo_PRD.bsdf_MIS_PDF = bsdf_sample.PDF;
    monte_carlo_PRD.path_PDF *= bsdf_sample.PDF;
    monte_carlo_PRD.clamped_path_PDF *= fminf(bsdf_sample.PDF, 1.0f);
    if (!is_PDF_valid(bsdf_sample.PDF))
        monte_carlo_PRD.throughput = make_float3(0.0f);
    else
        monte_carlo_PRD.throughput *= bsdf_sample.weight * (abs(bsdf_sample.direction.z) / bsdf_sample.PDF); // f * ||cos(theta)|| / pdf
    monte_carlo_PRD.bounces += 1u;
}

RT_PROGRAM void closest_hit() {
    // if (g_launch_index.x * 2 < g_accumulation_buffer.size().x)
    //     closest_hit_not_MIS();
    // else
        closest_hit_MIS();
}

//----------------------------------------------------------------------------
// Any hit program for monte carlo shadow rays.
//----------------------------------------------------------------------------

rtDeclareVariable(ShadowPRD, shadow_PRD, rtPayload, );

RT_PROGRAM void shadow_any_hit() {
    shadow_PRD.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

//=============================================================================
// Closest hit programs for monte carlo light sources.
//=============================================================================

RT_PROGRAM void light_closest_hit() {

    int light_index = __float_as_int(geometric_normal.x);
    const SphereLight& light = g_lights[light_index].sphere;

    float3 light_radiance = LightSources::evaluate(light, ray.origin, ray.direction);

    bool next_event_estimated = monte_carlo_PRD.bounces != 0; // Was next event estimated at previous intersection.
    bool apply_MIS = monte_carlo_PRD.bsdf_MIS_PDF > 0.0f;
    if (apply_MIS) {
        // Calculate MIS weight and scale the radiance by it.
        const float light_PDF = LightSources::PDF(light, ray.origin, ray.direction);
        float mis_weight = is_PDF_valid(light_PDF) ? RNG::power_heuristic(monte_carlo_PRD.bsdf_MIS_PDF, light_PDF) : 0.0f;
        light_radiance *= mis_weight;
    } else if (next_event_estimated)
        // Previous bounce used next event estimation, but did not calculate MIS, so don't apply light contribution.
        // TODO Could this be handled by setting bsdf_MIS_PDF to 0 instead? Wait until we have a specular BRDF implementation.
        light_radiance = make_float3(0.0f);

    float3 scaled_radiance = monte_carlo_PRD.throughput * light_radiance;
    monte_carlo_PRD.radiance += clamp_light_contribution_by_pdf(scaled_radiance, monte_carlo_PRD.clamped_path_PDF, g_accumulations);

    monte_carlo_PRD.throughput = make_float3(0.0f);
}