#include "hip/hip_runtime.h"
// OptiX path tracing ray generation and miss program.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Shading/ShadingModels/DefaultShading.h>
#include <OptiXRenderer/Shading/LightSources/SphereLightImpl.h>
#include <OptiXRenderer/TBN.h>
#include <OptiXRenderer/Types.h>

#include <optix.h>

using namespace optix;
using namespace OptiXRenderer;
using namespace OptiXRenderer::Shading::ShadingModels;

// Ray params
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(MonteCarloPRD, monte_carlo_PRD, rtPayload, );

// Scene params
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );
rtBuffer<SphereLight, 1> g_lights;
rtDeclareVariable(int, g_light_count, , );

// Material params
rtBuffer<Material, 1> g_materials;
rtDeclareVariable(int, material_index, , );

//----------------------------------------------------------------------------
// Closest hit program for monte carlo sampling rays.
//----------------------------------------------------------------------------

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

RT_PROGRAM void closest_hit() {
    // const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 forward_shading_normal = dot(world_shading_normal, -ray.direction) >= 0.0f ? world_shading_normal : -world_shading_normal;

    const TBN world_shading_tbn = TBN(forward_shading_normal);

    // Store intersection point and wo in PRD.
    monte_carlo_PRD.position = ray.direction * t_hit + ray.origin;
    monte_carlo_PRD.direction = world_shading_tbn * -ray.direction;

    const Material& material_parameter = g_materials[material_index];
    DefaultShading material = DefaultShading(material_parameter);

    // Sample light sources.
    for (int i = 0; i < g_light_count; ++i) {
        const SphereLight& light = g_lights[i];
        LightSample light_sample = LightSources::sample_radiance(light, monte_carlo_PRD.position, monte_carlo_PRD.rng.sample2f());
        float N_dot_L = dot(world_shading_tbn.get_normal(), light_sample.direction);
        light_sample.radiance *= abs(N_dot_L) / light_sample.PDF;

        // Inline the material response into the light sample's contribution.
        const float3 shading_light_direction = world_shading_tbn * light_sample.direction;
        const float3 bsdf_response = material.evaluate(monte_carlo_PRD.direction, shading_light_direction);// TODO Extend material and BRDFs with methods for evaluating contribution and PDF at the same time.
        light_sample.radiance *= bsdf_response;

        if (light_sample.radiance.x > 0.0f || light_sample.radiance.y > 0.0f || light_sample.radiance.z > 0.0f) {
            ShadowPRD shadow_PRD = { light_sample.radiance };
            Ray shadow_ray(monte_carlo_PRD.position, light_sample.direction, unsigned int(RayTypes::Shadow), g_scene_epsilon, light_sample.distance - g_scene_epsilon);
            rtTrace(g_scene_root, shadow_ray, shadow_PRD);

            monte_carlo_PRD.radiance += monte_carlo_PRD.throughput * shadow_PRD.attenuation;
        }
    }

    // Sample material.
    BSDFSample bsdf_sample = material.sample_all(monte_carlo_PRD.direction, monte_carlo_PRD.rng.sample3f());
    monte_carlo_PRD.direction = bsdf_sample.direction * world_shading_tbn;
    monte_carlo_PRD.bsdf_MIS_PDF = 0.0f; // bsdf_sample.PDF;
    monte_carlo_PRD.path_PDF *= bsdf_sample.PDF;
    if (!is_PDF_valid(bsdf_sample.PDF))
        monte_carlo_PRD.throughput = make_float3(0.0f);
    else
        monte_carlo_PRD.throughput *= bsdf_sample.weight * (abs(bsdf_sample.direction.z) / bsdf_sample.PDF); // f * ||cos(theta)|| / pdf
    monte_carlo_PRD.bounces += 1u;
}

//----------------------------------------------------------------------------
// Any hit program for monte carlo shadow rays.
//----------------------------------------------------------------------------

rtDeclareVariable(ShadowPRD, shadow_PRD, rtPayload, );

RT_PROGRAM void shadow_any_hit() {
    shadow_PRD.attenuation = make_float3(0.0f);
    rtTerminateRay();
}

//=============================================================================
// Closest hit programs for monte carlo light sources.
//=============================================================================

RT_PROGRAM void light_closest_hit() {

    if (monte_carlo_PRD.bounces == 0) {
        // This should only be sampled by rays leaving specular BRDFs right now!
        int light_index = __float_as_int(geometric_normal.x);
        const SphereLight& light = g_lights[light_index];

        monte_carlo_PRD.radiance += monte_carlo_PRD.throughput * LightSources::evaluate(light, ray.origin, ray.direction);
    }

    monte_carlo_PRD.throughput = make_float3(0.0f);
}