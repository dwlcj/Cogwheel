#include "hip/hip_runtime.h"
// OptiX path tracing ray generation and miss program.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Types.h>

#include <optix.h>

using namespace OptiXRenderer;
using namespace optix;

// Ray params
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(MonteCarloPRD, monte_carlo_PRD, rtPayload, );

// Material params
rtDeclareVariable(float3, g_color, , );

//----------------------------------------------------------------------------
// Closest hit program for monte carlo sampling rays.
//----------------------------------------------------------------------------

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

RT_PROGRAM void closest_hit() {
    const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    monte_carlo_PRD.color = g_color * abs(dot(world_geometric_normal, ray.direction));
}

//----------------------------------------------------------------------------
// Miss program for monte carlo rays.
//----------------------------------------------------------------------------

RT_PROGRAM void miss() {
    monte_carlo_PRD.color = make_float3(0.68f, 0.92f, 1.0f);
}