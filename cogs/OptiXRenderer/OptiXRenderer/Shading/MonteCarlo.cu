#include "hip/hip_runtime.h"
// OptiX path tracing ray generation program and integrator.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Types.h>

#include <optix.h>

using namespace OptiXRenderer;
using namespace optix;

rtDeclareVariable(float3, g_color, , );


rtDeclareVariable(MonteCarloPRD, monte_carlo_PRD, rtPayload, );

//----------------------------------------------------------------------------
// Closest hit program for monte carlo sampling rays.
//----------------------------------------------------------------------------

RT_PROGRAM void closest_hit() {
    monte_carlo_PRD.color = g_color;
}

//----------------------------------------------------------------------------
// Miss program for monte carlo rays.
//----------------------------------------------------------------------------

RT_PROGRAM void miss() {
    monte_carlo_PRD.color = make_float3(0.68f, 0.92f, 1.0f);
}