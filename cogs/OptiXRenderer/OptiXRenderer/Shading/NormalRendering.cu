#include "hip/hip_runtime.h"
// OptiX programs for visualizing a models normals.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Types.h>
#include <OptiXRenderer/Utils.h>

#include <optix.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace OptiXRenderer;
using namespace optix;

struct NormalVisualizationPayload {
    float4 color;
};

rtDeclareVariable(NormalVisualizationPayload, normal_visualization_payload, rtPayload, );

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );

rtDeclareVariable(float4, g_camera_position, , );
rtDeclareVariable(Matrix4x4, g_inverted_view_projection_matrix, , );

rtDeclareVariable(int, g_accumulations, , );
rtBuffer<ushort4, 2>  g_output_buffer;
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
rtBuffer<double4, 2>  g_accumulation_buffer;
#else
rtBuffer<float4, 2>  g_accumulation_buffer;
#endif

//----------------------------------------------------------------------------
// Ray generation program for visualizing normals.
//----------------------------------------------------------------------------
RT_PROGRAM void ray_generation() {
    if (g_accumulations == 0)
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
        g_accumulation_buffer[g_launch_index] = make_double4(0.0, 0.0, 0.0, 0.0);
#else
        g_accumulation_buffer[g_launch_index] = make_float4(0.0f);
#endif

    MonteCarloPayload payload;
    payload.radiance = make_float3(0.0f);
    payload.rng.seed(__brev(RNG::teschner_hash(g_launch_index.x, g_launch_index.y) ^ 83492791 ^ g_accumulations));
    payload.throughput = make_float3(1.0f);
    payload.bounces = 0;
    payload.bsdf_MIS_PDF = 0.0f;
    payload.shading_normal = make_float3(0.0f);

    // Generate rays.
    float2 screen_pos_offset = payload.rng.sample2f(); // Always advance the rng by two samples, even if we ignore them.
    float2 screen_pos = make_float2(g_launch_index) + (g_accumulations == 0 ? make_float2(0.5f) : screen_pos_offset);
    float2 viewport_pos = make_float2(screen_pos.x / float(g_accumulation_buffer.size().x), screen_pos.y / float(g_accumulation_buffer.size().y));
    payload.position = make_float3(g_camera_position);
    payload.direction = project_ray_direction(viewport_pos, payload.position, g_inverted_view_projection_matrix);

    // Iterate until a material is sampled.
    float3 last_ray_direction = payload.direction;
    do {
        last_ray_direction = payload.direction;
        Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene_epsilon);
        rtTrace(g_scene_root, ray, payload);
    } while (payload.bsdf_MIS_PDF == 0.0f && !is_black(payload.throughput));

    float D_dot_N = -dot(last_ray_direction, payload.shading_normal);
    if (D_dot_N < 0.0f)
        payload.radiance = make_float3(0.25f - 0.75f * D_dot_N, 0.0f, 0.0f);
    else
        payload.radiance = make_float3(0.0f, 0.25f + 0.75f * D_dot_N, 0.0f);

#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    double3 prev_radiance = make_double3(g_accumulation_buffer[g_launch_index].x, g_accumulation_buffer[g_launch_index].y, g_accumulation_buffer[g_launch_index].z);
    double3 accumulated_radiance_d = lerp_double(prev_radiance, make_double3(payload.radiance.x, payload.radiance.y, payload.radiance.z), 1.0 / (g_accumulations + 1.0));
    g_accumulation_buffer[g_launch_index] = make_double4(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z, 1.0f);
    float3 accumulated_radiance = make_float3(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z);
#else
    float3 prev_radiance = make_float3(g_accumulation_buffer[g_launch_index]);
    float3 accumulated_radiance = lerp(prev_radiance, payload.radiance, 1.0f / (g_accumulations + 1.0f));
    g_accumulation_buffer[g_launch_index] = make_float4(accumulated_radiance, 1.0f);
#endif

    g_output_buffer[g_launch_index] = float_to_half(make_float4(accumulated_radiance, 1.0f));
}