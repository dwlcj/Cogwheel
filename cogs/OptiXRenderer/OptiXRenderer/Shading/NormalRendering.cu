#include "hip/hip_runtime.h"
// OptiX path tracing ray generation program and integrator.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Types.h>

#include <optix.h>

struct NormalVisualizationPRD {
    float4 color;
};

rtDeclareVariable(NormalVisualizationPRD, normal_visualization_PRD, rtPayload, );

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );

rtBuffer<float4, 2>  g_accumulation_buffer; // TODO Make double4

//----------------------------------------------------------------------------
// Ray generation program for visualizing normals.
//----------------------------------------------------------------------------
RT_PROGRAM void ray_generation() {
    NormalVisualizationPRD prd;

    const float3 position = make_float3(0.0f, 0.0f, 0.0f);
    const float3 direction = make_float3(0.0f, 0.0f, 1.0f);

    //optix::Ray ray(position, direction, unsigned int(OptiXRenderer::RayTypes::NormalVisualization), g_scene_epsilon);
    //rtTrace(g_scene_root, ray, prd);

    g_accumulation_buffer[g_launch_index] = prd.color; 
}

//----------------------------------------------------------------------------
// Closest hit program for visualizing normals.
//----------------------------------------------------------------------------

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

RT_PROGRAM void closest_hit() {
    float3 remapped_normal = geometric_normal * 0.5f + 0.5f;
    normal_visualization_PRD.color = make_float4(remapped_normal, 1.0);
}

//----------------------------------------------------------------------------
// Miss program for normal visualization.
//----------------------------------------------------------------------------

RT_PROGRAM void miss() {
    normal_visualization_PRD.color = make_float4(0.0f, 0.0f, 0.0f, 1.0);
}