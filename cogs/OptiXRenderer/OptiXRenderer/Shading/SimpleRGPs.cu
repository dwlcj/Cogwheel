#include "hip/hip_runtime.h"
// Simple OptiX path ray generation programs, such as path tracing, normal and albedo visualization
// ------------------------------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License.
// See LICENSE.txt for more detail.
// ------------------------------------------------------------------------------------------------

#include <OptiXRenderer/Shading/ShadingModels/DefaultShading.h>
#include <OptiXRenderer/Shading/LightSources/LightImpl.h>
#include <OptiXRenderer/Types.h>
#include <OptiXRenderer/Utils.h>

#include <optix.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace OptiXRenderer;
using namespace optix;

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );

rtDeclareVariable(int, g_accumulations, , );
rtBuffer<ushort4, 2>  g_output_buffer;
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
rtBuffer<double4, 2>  g_accumulation_buffer;
#else
rtBuffer<float4, 2>  g_accumulation_buffer;
#endif

rtDeclareVariable(float4, g_camera_position, , );
rtDeclareVariable(Matrix4x4, g_inverted_view_projection_matrix, , );

// Scene variables
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );
rtDeclareVariable(int, g_max_bounce_count, , );

template <typename Evaluator>
__inline_dev__ void accumulate(Evaluator evaluator) {
    if (g_accumulations == 0)
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
        g_accumulation_buffer[g_launch_index] = make_double4(0.0, 0.0, 0.0, 0.0);
#else
        g_accumulation_buffer[g_launch_index] = make_float4(0.0f);
#endif

    MonteCarloPayload payload = initialize_monte_carlo_payload(g_launch_index.x, g_launch_index.y,
        g_accumulation_buffer.size().x, g_accumulation_buffer.size().y, g_accumulations,
        make_float3(g_camera_position), g_inverted_view_projection_matrix);

    float3 color = evaluator(payload);

#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    double3 prev_radiance = make_double3(g_accumulation_buffer[g_launch_index].x, g_accumulation_buffer[g_launch_index].y, g_accumulation_buffer[g_launch_index].z);
    double3 accumulated_radiance_d = lerp_double(prev_radiance, make_double3(color.x, color.y, color.z), 1.0 / (g_accumulations + 1.0));
    g_accumulation_buffer[g_launch_index] = make_double4(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z, 1.0f);
    float3 accumulated_radiance = make_float3(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z);
#else
    float3 prev_radiance = make_float3(g_accumulation_buffer[g_launch_index]);
    float3 accumulated_radiance = lerp(prev_radiance, color, 1.0f / (g_accumulations + 1.0f));
    g_accumulation_buffer[g_launch_index] = make_float4(accumulated_radiance, 1.0f);
#endif

    g_output_buffer[g_launch_index] = float_to_half(make_float4(accumulated_radiance, 1.0f));
}

//-------------------------------------------------------------------------------------------------
// Path tracing ray generation program.
//-------------------------------------------------------------------------------------------------
RT_PROGRAM void path_tracing_RPG() {

    accumulate([](MonteCarloPayload payload) -> float3 {
        do {
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene_epsilon);
            rtTrace(g_scene_root, ray, payload);
        } while (payload.bounces < g_max_bounce_count && !is_black(payload.throughput));

        return payload.radiance;
    });
}

//-------------------------------------------------------------------------------------------------
// Ray generation program for visualizing normals.
//-------------------------------------------------------------------------------------------------
RT_PROGRAM void normals_RPG() {

    accumulate([](MonteCarloPayload payload) -> float3 {
        // Iterate until a material is sampled.
        float3 last_ray_direction = payload.direction;
        do {
            last_ray_direction = payload.direction;
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene_epsilon);
            rtTrace(g_scene_root, ray, payload);
        } while (payload.bsdf_MIS_PDF == 0.0f && !is_black(payload.throughput));

        float D_dot_N = -dot(last_ray_direction, payload.shading_normal);
        if (D_dot_N < 0.0f)
            return make_float3(0.25f - 0.75f * D_dot_N, 0.0f, 0.0f);
        else
            return make_float3(0.0f, 0.25f + 0.75f * D_dot_N, 0.0f);
    });
}

//-------------------------------------------------------------------------------------------------
// Ray generation program for visualizing estimated and sampled albedo.
//-------------------------------------------------------------------------------------------------
rtBuffer<Material, 1> g_materials;

RT_PROGRAM void albedo_RPG() {

    accumulate([](MonteCarloPayload payload) -> float3 {
        float3 last_ray_direction = payload.direction;
        do {
            last_ray_direction = payload.direction;
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene_epsilon);
            rtTrace(g_scene_root, ray, payload);
        } while (payload.material_index == 0 && !is_black(payload.throughput));

        bool valid_material = payload.material_index != 0;
        if (g_launch_index.x < g_accumulation_buffer.size().x / 2 && valid_material) {
            using namespace Shading::ShadingModels;
            const Material& material_parameter = g_materials[payload.material_index];
            const DefaultShading material = DefaultShading(material_parameter, payload.texcoord);
            return material.IBL(last_ray_direction, payload.shading_normal, 0); // TODO Use a white IBL.
        } else
            return payload.throughput;
    });
}

//-------------------------------------------------------------------------------------------------
// Miss program for monte carlo rays.
//-------------------------------------------------------------------------------------------------

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(MonteCarloPayload, monte_carlo_payload, rtPayload, );
rtDeclareVariable(float3, g_scene_environment_tint, , );
#if PRESAMPLE_ENVIRONMENT_MAP
rtDeclareVariable(PresampledEnvironmentLight, g_scene_environment_light, , );
#else
rtDeclareVariable(EnvironmentLight, g_scene_environment_light, , );
#endif

RT_PROGRAM void miss() {
    float3 environment_radiance = g_scene_environment_tint;

    unsigned int environment_map_ID = g_scene_environment_light.environment_map_ID;
    if (environment_map_ID) {
        bool next_event_estimated = monte_carlo_payload.bounces != 0; // Was next event estimated at previous intersection.
        environment_radiance *= LightSources::evaluate_intersection(g_scene_environment_light, ray.origin, ray.direction, 
                                                                    monte_carlo_payload.bsdf_MIS_PDF, next_event_estimated);
    }

    monte_carlo_payload.radiance += monte_carlo_payload.throughput * environment_radiance;
    monte_carlo_payload.throughput = make_float3(0.0f);
    monte_carlo_payload.shading_normal = -ray.direction;
}

//-------------------------------------------------------------------------------------------------
// Exception program.
//-------------------------------------------------------------------------------------------------
RT_PROGRAM void exceptions() {
    rtPrintExceptionDetails();

#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    g_accumulation_buffer[g_launch_index] = make_double4(100000, 0, 0, 1.0);
#else
    g_accumulation_buffer[g_launch_index] = make_float4(100000, 0, 0, 1.0f);
#endif
}
