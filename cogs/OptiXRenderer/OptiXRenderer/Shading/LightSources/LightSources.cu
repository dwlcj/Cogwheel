#include "hip/hip_runtime.h"
// OptiX light source intersection programs.
// ---------------------------------------------------------------------------
// Copyright (C) 2015-2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Intersect.h>
#include <OptiXRenderer/Shading/LightSources/SphereLightImpl.h>

#include <optix.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_math.h>

using namespace OptiXRenderer;
using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtBuffer<Light, 1> g_lights;

// Encode light index in geometric_normal.x
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

//=============================================================================
// Sphere light intersection programs.
//=============================================================================
RT_PROGRAM void intersect(int primitive_index) {

    // Only sphere lights can be intersected.
    if (g_lights[primitive_index].get_type() != Light::Sphere) return;

    const SphereLight& light = g_lights[primitive_index].sphere;

    float t = Intersect::ray_sphere(ray, Sphere::make(light.position, light.radius));
    if (t > 0.0f && rtPotentialIntersection(t)) {
        float3 intersection_point = t * ray.direction + ray.origin;
        float inv_radius = 1.0f / light.radius;
        shading_normal = (intersection_point - light.position) * inv_radius;
        geometric_normal.x = __int_as_float(primitive_index);
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bounds(int primitive_index, float result[6]) {
    optix::Aabb* aabb = (optix::Aabb*)result;

    if (g_lights[primitive_index].get_type() != Light::Sphere) {
        aabb->invalidate();
        return;
    }

    const SphereLight& light = g_lights[primitive_index].sphere;

    if (light.radius > 0.0f) {
        aabb->m_min = light.position - light.radius;
        aabb->m_max = light.position + light.radius;
    } else
        aabb->invalidate();
}