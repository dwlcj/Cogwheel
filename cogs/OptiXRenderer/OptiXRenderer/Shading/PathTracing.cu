#include "hip/hip_runtime.h"
// OptiX path tracing ray generation program and integrator.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Shading/Utils.h>
#include <OptiXRenderer/Types.h>

#include <optix.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace OptiXRenderer;
using namespace OptiXRenderer::Shading;
using namespace optix;

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );

rtDeclareVariable(float, g_accumulations, , );
rtBuffer<float4, 2>  g_accumulation_buffer;

rtDeclareVariable(float4, g_camera_position, , );
rtDeclareVariable(Matrix4x4, g_inverted_view_projection_matrix, , );

rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );

//----------------------------------------------------------------------------
// Ray generation program
//----------------------------------------------------------------------------
RT_PROGRAM void path_tracing() {
    if (g_accumulations == 0.0f)
        g_accumulation_buffer[g_launch_index] = make_float4(0.0, 0.0, 0.0, 0.0);

    // Generate rays.
    float2 viewport_pos = make_float2(g_launch_index.x / float(g_accumulation_buffer.size().x), g_launch_index.y / float(g_accumulation_buffer.size().y));
    float3 origin = make_float3(g_camera_position);
    float3 direction = project_ray_direction(viewport_pos, origin, g_inverted_view_projection_matrix);
    Ray ray(origin, direction, unsigned int(RayTypes::MonteCarlo), g_scene_epsilon);

    MonteCarloPRD prd;
    rtTrace(g_scene_root, ray, prd);

    // Simple gamma correction.
    float inv_screen_gamma = 1.0f / 2.2f;
    prd.color = gammacorrect(prd.color, inv_screen_gamma);

    g_accumulation_buffer[g_launch_index] = make_float4(prd.color, 1.0f);
}
