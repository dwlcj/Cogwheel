#include "hip/hip_runtime.h"
// OptiX path tracing ray generation program and integrator.
// ---------------------------------------------------------------------------
// Copyright (C) 2016, Cogwheel. See AUTHORS.txt for authors
//
// This program is open source and distributed under the New BSD License. See
// LICENSE.txt for more detail.
// ---------------------------------------------------------------------------

#include <OptiXRenderer/Shading/LightSources/EnvironmentLightImpl.h>
#include <OptiXRenderer/Types.h>
#include <OptiXRenderer/Utils.h>

#include <optix.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace OptiXRenderer;
using namespace optix;

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );

rtDeclareVariable(int, g_accumulations, , );
rtBuffer<float4, 2>  g_output_buffer;
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
rtBuffer<double4, 2>  g_accumulation_buffer;
#else
rtBuffer<float4, 2>  g_accumulation_buffer;
#endif

rtDeclareVariable(float4, g_camera_position, , );
rtDeclareVariable(Matrix4x4, g_inverted_view_projection_matrix, , );

// Scene variables
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(float, g_scene_epsilon, , );
rtDeclareVariable(int, g_max_bounce_count, , );

__inline_dev__ bool is_black(const optix::float3 color) {
    return color.x <= 0.0f && color.y <= 0.0f && color.z <= 0.0f;
}

__inline_dev__ inline optix::double3 lerp_double(const optix::double3& a, const optix::double3& b, const double t) {
    return optix::make_double3(a.x + (b.x - a.x) * t, a.y + (b.y - a.y) * t, a.z + (b.z - a.z) * t);
}

//----------------------------------------------------------------------------
// Ray generation program
//----------------------------------------------------------------------------
RT_PROGRAM void path_tracing() {
    if (g_accumulations == 0)
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
        g_accumulation_buffer[g_launch_index] = make_double4(0.0, 0.0, 0.0, 0.0);
#else
        g_accumulation_buffer[g_launch_index] = make_float4(0.0f);
#endif

    unsigned int index = g_launch_index.y * g_accumulation_buffer.size().x + g_launch_index.x;

    MonteCarloPayload payload;
    payload.radiance = make_float3(0.0f);
    // payload.rng.seed(RNG::hash(index) ^ __brev(g_accumulations));
    // payload.rng.seed(__brev(g_accumulations)); // Uniform seed.
    payload.rng.seed(__brev(morton_encode(g_launch_index.x, g_launch_index.y)) ^ 674506081 * g_accumulations);
    payload.throughput = make_float3(1.0f);
    payload.bounces = 0;
    payload.bsdf_MIS_PDF = 0.0f;
    payload.clamped_path_PDF = payload.path_PDF = 1.0f;

    // Generate rays.
    float2 screen_pos_offset = payload.rng.sample2f(); // Always advance the rng by two samples, even if we ignore them.
    float2 screen_pos = make_float2(g_launch_index) + (g_accumulations == 0 ? make_float2(0.5f) : screen_pos_offset);
    float2 viewport_pos = make_float2(screen_pos.x / float(g_accumulation_buffer.size().x), screen_pos.y / float(g_accumulation_buffer.size().y));
    payload.position = make_float3(g_camera_position);
    payload.direction = project_ray_direction(viewport_pos, payload.position, g_inverted_view_projection_matrix);

    do {
        Ray ray(payload.position, payload.direction, unsigned int(RayTypes::MonteCarlo), g_scene_epsilon);
        rtTrace(g_scene_root, ray, payload);
    } while (payload.bounces < g_max_bounce_count && !is_black(payload.throughput));

#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    double3 prev_radiance = make_double3(g_accumulation_buffer[g_launch_index].x, g_accumulation_buffer[g_launch_index].y, g_accumulation_buffer[g_launch_index].z);
    double3 accumulated_radiance_d = lerp_double(prev_radiance, make_double3(payload.radiance.x, payload.radiance.y, payload.radiance.z), 1.0 / (g_accumulations + 1.0));
    g_accumulation_buffer[g_launch_index] = make_double4(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z, 1.0f);
    float3 accumulated_radiance = make_float3(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z);
#else
    float3 prev_radiance = make_float3(g_accumulation_buffer[g_launch_index]);
    float3 accumulated_radiance = lerp(prev_radiance, payload.radiance, 1.0f / (g_accumulations + 1.0f));
    g_accumulation_buffer[g_launch_index] = make_float4(accumulated_radiance, 1.0f);
#endif

    // Apply simple gamma correction to the output.
    const float inv_screen_gamma = 1.0f / 2.2f;
    g_output_buffer[g_launch_index] = make_float4(gammacorrect(accumulated_radiance, inv_screen_gamma), 1.0f);
}

//----------------------------------------------------------------------------
// Miss program for monte carlo rays.
//----------------------------------------------------------------------------

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(MonteCarloPayload, monte_carlo_payload, rtPayload, );
rtDeclareVariable(float3, g_scene_environment_tint, , );
rtDeclareVariable(EnvironmentLight, g_scene_environment_light, , );

RT_PROGRAM void miss() {
    float3 environment_radiance = g_scene_environment_tint;

    unsigned int environment_map_ID = g_scene_environment_light.environment_map_ID;
    if (environment_map_ID) {
        environment_radiance *= LightSources::evaluate(g_scene_environment_light, ray.origin, ray.direction);
        
        // NOTE We can get rid of all these branches by just scaling the (mis) weight. Requires a lot of retesting though. :)
        bool next_event_estimatable = g_scene_environment_light.per_pixel_PDF_ID != RT_TEXTURE_ID_NULL;
        if (next_event_estimatable) {
            bool next_event_estimated = monte_carlo_payload.bounces != 0; // Was next event estimated at previous intersection.
            bool apply_MIS = monte_carlo_payload.bsdf_MIS_PDF > 0.0f;
            if (apply_MIS) {
                // Calculate MIS weight and scale the radiance by it.
                const float light_PDF = LightSources::PDF(g_scene_environment_light, ray.origin, ray.direction);
                float mis_weight = RNG::power_heuristic(monte_carlo_payload.bsdf_MIS_PDF, light_PDF);
                environment_radiance *= mis_weight;
            } else if (next_event_estimated)
                // Previous bounce used next event estimation, but did not calculate MIS, so don't apply light contribution.
                // TODO Could this be handled by setting bsdf_MIS_PDF to 0 instead? 
                //      Wait until we have a specular BRDF implementation and
                //      remember to test with next event estimation on and off.
                environment_radiance = make_float3(0.0f);
        }
    }

    float3 scaled_radiance = clamp_light_contribution_by_path_PDF(environment_radiance, monte_carlo_payload.clamped_path_PDF, g_accumulations);
    monte_carlo_payload.radiance += monte_carlo_payload.throughput * scaled_radiance;

    monte_carlo_payload.throughput = make_float3(0.0f);
}

//----------------------------------------------------------------------------
// Exception program.
//----------------------------------------------------------------------------
RT_PROGRAM void exceptions() {
    rtPrintExceptionDetails();

#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    g_accumulation_buffer[g_launch_index] = make_double4(100000, 0, 0, 1.0);
#else
    g_accumulation_buffer[g_launch_index] = make_float4(100000, 0, 0, 1.0f);
#endif
}
